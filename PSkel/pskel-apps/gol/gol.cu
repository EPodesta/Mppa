#include <omp.h>
#include <fstream>
#include <string>
#include <stdio.h>
#include <iostream>
#include <sstream>

#include <unistd.h>

#include <cassert>

//#define PSKEL_SHARED_MASK
#include "../include/PSkel.h"

#include "../util/hr_time.h"

using namespace std;
using namespace PSkel;

namespace PSkel{
	__parallel__ void stencilKernel(Array2D<int> input,Array2D<int> output,Mask2D<int> mask,int null, size_t h, size_t w){
		int neighbors=0;
		for(int z=0;z<mask.size;z++){
			neighbors += mask.get(z,input,h,w);
		}	
		output(h,w) = ((neighbors==3 || (input(h,w)==1 && neighbors==2))?1:0);
	}
}

int main(int argc, char **argv){
	int width,height,iterations,GPUBlockSize,numCPUThreads,mode,tileHeight,tileIterations;
	if (argc != 9){
		printf ("Wrong number of parameters.\n", argv[0]);
		printf ("Usage: gol WIDTH HEIGHT ITERATIONS MODE GPUBLOCKS CPUTHREADS TILEHEIGHT TILEITERATIONS\n");
		exit (-1);
	}

	width = atoi (argv[1]);
	height = atoi (argv[2]);
	iterations = atoi (argv[3]);
	mode = atoi(argv[4]);
	GPUBlockSize = atoi(argv[5]);
	numCPUThreads = atoi(argv[6]);
	tileHeight = atoi(argv[7]);
	tileIterations = atoi(argv[8]);
	
	Array2D<int> inputGrid(width,height);
	Array2D<int> outputGrid(width,height);
	
	Mask2D<int> mask(8);
	mask.set(0,-1,-1);	mask.set(1,-1,0);	mask.set(2,-1,1);
	mask.set(3,0,-1);				mask.set(4,0,1);
	mask.set(5,1,-1);	mask.set(6,1,0);	mask.set(7,1,1);
	
	srand(123456789);
	for(int h=0;h<height;h++) {
		for(int w=0;w<width;w++) {
			inputGrid(h,w) = rand()%2;
		}
	}
	
	Stencil2D<Array2D<int>, Mask2D<int>, int> stencil(inputGrid, outputGrid, mask, 0);
	hr_timer_t timer;
	switch(mode){
	case 0:
		hrt_start(&timer);
		stencil.runIterativeSequential(iterations);
		hrt_stop(&timer);
		break;
	case 1:
		hrt_start(&timer);
		stencil.runIterativeCPU(iterations,numCPUThreads);
		hrt_stop(&timer);
		break;
	case 2:
		hrt_start(&timer);
		stencil.runIterativeGPU(iterations, GPUBlockSize);	
		hrt_stop(&timer);
		break;
	case 3:
		hrt_start(&timer);
		stencil.runIterativeTilingGPU(iterations, width, tileHeight, 1, tileIterations, GPUBlockSize);	
		hrt_stop(&timer);
		break;
	case 4:
		hrt_start(&timer);
		stencil.runIterativeAutoGPU(iterations, GPUBlockSize);	
		hrt_stop(&timer);
		break;
	}
	cout << hrt_elapsed_time(&timer);
	inputGrid.hostFree();
	outputGrid.hostFree();
	return 0;
}
